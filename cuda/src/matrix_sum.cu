
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

/*
// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        cudaError_t __err = cudaGetLastError(); \
        if (__err != cudaSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, cudaGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const size_t DSIZE = 32768;      // matrix side dimension
const int BLOCK_SIZE = 256;      // CUDA maximum is 1024

// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t n){

  int idx = threadIdx.x + blockIdx.x * blockDim.x; // create typical 1D thread index from built-in variables
  if (idx < n){
    float sum = 0.0f;
    for (size_t i = 0; i < n; i++)
      sum += A[idx * DSIZE + i];         // write a for loop that will cause the thread to iterate across a row, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
  }
}


__device__ inline float WarpSum(float val)
{
    const uint mask = 0xFFFFFFFF;
    for (uint k = warpSize / 2; k > 0; k >>= 1) {
        val += __shfl_down_sync(mask, val, k);
    }
    return val;
}


// there are n blocks, each of size BS (may be < n)
#define BS 1024
__global__ void row_sums_reduce(const float* A, float* sums, size_t n)
{
    __shared__ float block[32];
    const uint y = blockIdx.x;
    const uint lane = threadIdx.x % warpSize;
    const uint warp = threadIdx.x / warpSize;
    const uint warpCount = blockDim.x / warpSize;

    // Our thread block will sum the row at blockIdx.x
    float val = 0;
    for (uint xBase = 0; xBase < n; xBase += blockDim.x) {
        // load the block
        if (xBase + threadIdx.x < n) {
            val += A[y * n + xBase + threadIdx.x];
        }
    }

    val = WarpSum(val);
    if (lane == 0) {
        block[warp] = val;
    }
    __syncthreads();

    if (warp == 0) {
        val = lane < warpCount ? block[lane] : 0;
        val = WarpSum(val);
        if (lane == 0) {
            sums[y] = val;
        }  
    }
}
#undef BS


// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x + blockIdx.x * blockDim.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[i * DSIZE + idx];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}

// blocks of size (BS, BS)
// grid of size (GS, 1) with BS*GS >= n
#define BS 32
__global__ void column_sums_blocks(const float* A, float* sums, size_t n)
{
    __shared__ float block[BS][BS+1];
    const uint tx = threadIdx.x;
    const uint ty = threadIdx.y;
    const uint xBlock = blockDim.x * blockIdx.x;
    assert(xBlock < n);
    assert(warpSize == BS);

    // Accumulate the values of the block
    float val = 0;
    for (uint yBlock = blockDim.y * blockIdx.y; yBlock < n; yBlock += blockDim.y) {
        if (xBlock + tx < n && yBlock + ty < n) {
            val += A[(xBlock + tx) + n * (yBlock + ty)];
        }
    }
    // transpose the values of the block
    block[ty][tx] = val;
    __syncthreads();
    val = block[tx][ty];

    // each warp sums its values
    const uint mask = 0xFFFFFFFF;
    for (uint k = warpSize / 2; k > 0; k >>= 1) {
        val += __shfl_down_sync(mask, val, k);
    }
    if (tx == 0) {
        sums[xBlock + ty] = val;
    }
}
#undef BS



bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}

int main(){

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  h_sums = new float[DSIZE]();
    
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
    
  cudaMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  cudaMalloc(&d_sums, DSIZE*sizeof(float)); // allocate device space for vector d_sums
  cudaCheckErrors("cudaMalloc failure"); // error checking
    
  // copy matrix A to device:
  cudaMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), cudaMemcpyHostToDevice);
  cudaCheckErrors("cudaMemcpy H2D failure");

  {  
    //cuda processing sequence step 1 is complete
    row_sums<<<(DSIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sums, DSIZE);
    cudaCheckErrors("kernel launch failure");
    //cuda processing sequence step 2 is complete
        
    // copy vector sums from device to host:
    cudaMemcpy(h_sums, d_sums, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
        
    //cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or cudaMemcpy H2D failure");
        
    if (!validate(h_sums, DSIZE)) return -1; 
    printf("row sums correct!\n");
  }  

  {  
    cudaMemset(d_sums, 0, DSIZE*sizeof(float));
        
    column_sums<<<(DSIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sums, DSIZE);
    cudaCheckErrors("kernel launch failure");
    // cuda processing sequence step 2 is complete
        
    // copy vector sums from device to host:
    cudaMemcpy(h_sums, d_sums, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
    //cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or cudaMemcpy H2D failure");
        
    if (!validate(h_sums, DSIZE)) return -1; 
    printf("column sums correct!\n");
  }

  {     
    cudaMemset(d_sums, 0, DSIZE*sizeof(float));
        
    dim3 block_num((DSIZE + 32 - 1) / 32, 1);
    dim3 block_size(32, 32);
    column_sums_blocks<<<block_num, block_size>>>(d_A, d_sums, DSIZE);
    cudaCheckErrors("kernel launch failure");
    //cuda processing sequence step 2 is complete
        
    // copy vector sums from device to host:
    cudaMemcpy(h_sums, d_sums, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
    //cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or cudaMemcpy H2D failure");
        
    if (!validate(h_sums, DSIZE)) return -1; 
    printf("column sums correct!\n");
  }

  {  
    cudaMemset(d_sums, 0, DSIZE*sizeof(float));
        
    row_sums_reduce<<<DSIZE, 1024>>>(d_A, d_sums, DSIZE);
    cudaCheckErrors("kernel launch failure");
    //cuda processing sequence step 2 is complete
        
    // copy vector sums from device to host:
    cudaMemcpy(h_sums, d_sums, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
    //cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or cudaMemcpy H2D failure");
        
    if (!validate(h_sums, DSIZE)) return -1; 
    printf("row sums correct!\n");
  }

  return 0;
}*/
  